#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <opencv\cv.h>
#include <opencv\highgui.h>

/*
extern "C" void draw(float *pcuSrc, float *pcuDst, int w, int h, float *cuGkernel, int kernel_size);
extern "C" __global__ void drawline(float *line , float *dot)
{

}
struct Point {
	float x;
	float y;
};
extern __host__ __device__ int		   abs(int) ;
extern __host__ __device__ float		   cos(int);
extern __device__ float sqrt(int);
__constant__ int point[8];
__global__ void historgram(float *buf, int x, int y ,unsigned short *histo)
{

}
__device__ Point distance(float x1,float y1, float x2, float y2,float x,float y)
{
	// get dot product of e1, e2
	int _ex1 = x2 - x1;
	int _ey1 = y2 - y1;
	int _ex2 = x - x1;
	int _ey2 = y - y1;
	float val = _ex1*_ex2 + _ey1*_ey2;
	float len = _ex1*_ex1 + _ey1*_ey1;
	float r1 = x1 + (val*_ex1) / len;
	float r2 = y1 + (val*_ey1) / len;
	//float dot = (float)(_ex1*_ex2 + _ey1*_ey2);
	//float len1 = sqrtf(_ex1*_ex1 + _ey1*_ey1);
	//float len2 = sqrtf(_ex2*_ex2 + _ey2*_ey2);
	//float cos = dot / (len1*len2);
	//float proj = cos*len2;
	//float r1 = x1 + (proj*_ex1) / len1;
	//float r2 = y1 + (proj*_ey1) / len1;
	Point ret;
	ret.x = r1;
	ret.y = r2;
	return ret;
}

__device__ bool isOnLine(float x1,float y1,float x2,float y2,float x, float y)
{
	Point e1;
	e1.x = x2 - x1;
	e1.y = y2 - y1;

	float rec = e1.x*e1.x + e1.y*e1.y;
	Point e2;
	e2.x = x - x1;
	e2.y = y - y1;
	float val = e1.x*e2.x + e1.y*e2.y;
	return (val >= 0 && val <= rec);

}
__device__ int cross2D(int x1,int y1,int x2,int y2)
{
	return x1*y2 - y1*x2;
}
__device__ int distance2D(int _x, int _y, int _x1, int _y1)
{
	
	return abs(cross2D(_x, _y, _x1, _y1)*cross2D(_x, _y, _x1, _y1)) / (_x1*_x1 + _y1*_y1);
}
__global__ void cuda_Filter2D(float * pSrcImage, int SrcWidth, int SrcHeight, float *pKernel, int KWidth, int KHeight, float *pDstImage)
{
	
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int index = y*SrcWidth + x;
	int border;
	float tempX,tempY,magnitude;
	bool a = true;

	int weightX[9] = {  -1,  0,  1 , -2,  0,  2 ,-1,  0,  1  };
	int weightY[9] = { 1,2,1,0,0,0,-1,-2,-1 };

	if (x>3 / 2 && y>3 / 2 && x < SrcWidth - 3 / 2 && y < SrcHeight - 3 / 2)
	{

		tempX = 0; tempY = 0;
		// make cuda code here !!!!!!!!!!!!!!!!!!!!!!!!!
		for (int i = 0; i < 3; i++)
		{
			for (int j = 0; j < 3; j++)
			{
				border = (y + i)*SrcWidth + (x + j);
				tempX += (pSrcImage[border] * weightX[i*3 + j]);
				tempY += (pSrcImage[border] * weightY[i * 3 + j]);

			}
		}
		magnitude = sqrt(tempX*tempX + tempY*tempY)+0.5;
		
		pDstImage[index] = magnitude > 120 ? 2 : 255;
	}


	//int ret;
	//int x1 = pKernel[0];
	//int y1 = pKernel[1];
	//int x2 = pKernel[2];
	//int y2 = pKernel[3];


	//if (isOnLine(x1, y1, x2, y2, x, y)) {
	//	Point ret = distance(x1, y1, x2, y2, x, y);
	//	if ( ((ret.x-x)* (ret.x - x)+ (ret.y - y)* (ret.y - y)) < 100000) {
	//		int index_ret = (int)(ret.y*(float)SrcWidth + ret.x);
	//		pDstImage[index_ret] = 127;

	//		//if (abs(pSrcImage[index_ret] - pSrcImage[index])<=50)
	//			//pDstImage[index] = pSrcImage[index];
	//		//else
	//			//pDstImage[index] = 127;
	//	}
	//	else {
	//		pDstImage[index] = 127;

	//	}
	//}
	//else
	//{
	//	pDstImage[index] = pSrcImage[index];
	//}




}
__global__ void cuda_lowpass2D(float * pSrcImage, int SrcWidth, int SrcHeight, float *pKernel, int KWidth, int KHeight, float *pDstImage)
{
	float Kernel[25] = { 1 / 25.0,  1 / 25.0,  1 / 25.0,  1 / 25.0,  1 / 25.0, 
		1 / 25.0,  1 / 25.0,  1 / 25.0,  1 / 25.0,  1 / 25.0, 
		1 / 25.0,  1 / 25.0,  1 / 25.0,  1 / 25.0,  1 / 25.0, 
		1 / 25.0,  1 / 25.0,  1 / 25.0,  1 / 25.0,  1 / 25.0, 
		1 / 25.0,  1 / 25.0,  1 / 25.0,  1 / 25.0,  1 / 25.0 };
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int index = y*SrcWidth + x;
	int border;
	float tempX, tempY, magnitude;
	if (x>5 / 2 && y>5 / 2 && x < SrcWidth - 5 / 2 && y < SrcHeight - 5 / 2)
	{

		tempX = 0; tempY = 0;
		// make cuda code here !!!!!!!!!!!!!!!!!!!!!!!!!
		for (int i = 0; i < 5; i++)
		{
			for (int j = 0; j < 5; j++)
			{
				border = (y + i)*SrcWidth + (x + j);
				tempX += (pSrcImage[border] * Kernel[i * 5 + j]);

			}
		}
		pDstImage[index] = tempX;
	}
	else {
		pDstImage[index] = pSrcImage[index];
	}
}
__global__ void line(int x0, int y0, int x1, int y1) {

	int dx = abs(x1 - x0), sx = x0<x1 ? 1 : -1;
	int dy = abs(y1 - y0), sy = y0<y1 ? 1 : -1;
	int err = (dx>dy ? dx : -dy) / 2, e2;

	for (;;) {
		
		if (x0 == x1 && y0 == y1) break;
		e2 = err;
		if (e2 >-dx) { err -= dy; x0 += sx; }
		if (e2 < dy) { err += dx; y0 += sy; }
	}
}
__global__ void cuda_full(float* pSrcImage, int w, int h, float *pKernel,float* pDstImage)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int index = y*w + x;


	int ret;
	int x1 = pKernel[0];
	int y1 = pKernel[1];
	int x2 = pKernel[2];
	int y2 = pKernel[3];
	bool k = false;
	short count = 0;
	if (isOnLine(x1, y1, x2, y2, x, y)) {
		Point ret = distance(x1, y1, x2, y2, x, y);
		int x0 = (int)ret.x;
		int y0 = (int)ret.y;
		if ( ((x0-x)* (x0 - x)+ (y0 - y)* (y0 - y)) < 100000) {
			int dx = abs(x - x0), sx = x0 < x ? 1 : -1;
			int dy = abs(y - y0), sy = y0 < y ? 1 : -1;
			int err = (dx>dy ? dx : -dy) / 2, e2;

			for (;;) {
				if (pSrcImage[y0*w + x0] == 2 )
				{
					k = true;
					break;
				}
				if (x0 == x && y0 == y) break;
				e2 = err;
				if (e2 >-dx) { err -= dy; x0 += sx; }
				if (e2 < dy) { err += dx; y0 += sy; }
			}
			pDstImage[index] =2;
		}
		else {
			pDstImage[index] = pSrcImage[index];

		}
	}
	else
	{
		pDstImage[index] = pSrcImage[index];
	}
	if(k)
		pDstImage[index] = pSrcImage[index];



}


void draw(float *pcuSrc, float *pcuDst, int w, int h, float *cuGkernel, int kernel_size)
{
	dim3 block = dim3(32, 32, 1);
	dim3 grid = dim3(w / block.x, h / block.y);

	cuda_Filter2D <<<grid, block >>> (pcuSrc, w, h, cuGkernel, kernel_size, kernel_size, pcuDst);
	cuda_full << <grid, block >> > (pcuDst, w, h,cuGkernel,pcuSrc);
	hipDeviceSynchronize();

}
*/

extern "C" void sub(float**pcuSrc, int w, int h, float *pcuDst, float *bmask, int *pPoint, float *temp);

extern "C" __global__ void drawline(float *line, float *dot)
{

}
struct Point {
	float x;
	float y;
};
extern __host__ __device__ int         abs(int);
extern __host__ __device__ float         cos(int);
extern __device__ float sqrt(int);
extern __device__ float sqrtf(float);

__constant__ int point[8];
__global__ void historgram(float *buf, int x, int y, unsigned short *histo)
{

}
__device__ Point distance(float x1, float y1, float x2, float y2, float x, float y)
{
	// get dot product of e1, e2
	int _ex1 = x2 - x1;
	int _ey1 = y2 - y1;
	int _ex2 = x - x1;
	int _ey2 = y - y1;
	float val = _ex1*_ex2 + _ey1*_ey2;
	float len = _ex1*_ex1 + _ey1*_ey1;

	float r1 = x1 + (val*_ex1) / len;
	float r2 = y1 + (val*_ey1) / len;

	Point ret;
	ret.x = r1;
	ret.y = r2;
	return ret;
}

__device__ bool isOnLine(float x1, float y1, float x2, float y2, float x, float y)
{
	Point e1;
	e1.x = x2 - x1;
	e1.y = y2 - y1;

	float rec = e1.x*e1.x + e1.y*e1.y;
	Point e2;
	e2.x = x - x1;
	e2.y = y - y1;
	float val = e1.x*e2.x + e1.y*e2.y;
	return (val >= 0 && val <= rec);

}

__global__ void cuda_low2D(float * src0, float * src1, float * src2, float * src3, float * src4, float * src5, float * src6, float * src7, float * src8, float * src9, int w, int h, float *pDstImage)
{

	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int index = y*w + x;
	int border;
	int count = 0;
	float temp[10] = { 0 };
	if (x>3 / 2 && y>3 / 2 && x < w - 3 / 2 && y < h - 3 / 2)
	{

		for (int i = 0; i < 3; i++)
		{
			for (int j = 0; j < 3; j++)
			{
				border = (y + i)*w + (x + j);
				temp[0] += src0[border] * 1 / 9.0;
				temp[1] += src1[border] * 1 / 9.0;
				temp[2] += src2[border] * 1 / 9.0;
				temp[3] += src3[border] * 1 / 9.0;
				temp[4] += src4[border] * 1 / 9.0;
				temp[5] += src5[border] * 1 / 9.0;
				temp[6] += src6[border] * 1 / 9.0;
				temp[7] += src7[border] * 1 / 9.0;
				temp[8] += src8[border] * 1 / 9.0;
				temp[9] += src9[border] * 1 / 9.0;

			}
		}
		src0[index] = temp[0];
		src1[index] = temp[1];
		src2[index] = temp[2];
		src3[index] = temp[3];
		src4[index] = temp[4];
		src5[index] = temp[5];
		src6[index] = temp[6];
		src7[index] = temp[7];
		src8[index] = temp[8];
		src9[index] = temp[9];


		pDstImage[index] = temp[4];
	}
	else {
		pDstImage[index] = src4[0];
	}

}
__global__ void cuda_back(float * src0, float * src1, float * src2, float * src3, float * src4, float * src5, float * src6, float * src7, float * src8, float * src9, int w, int h, float *pDstImage)
{

	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int index = y*w + x;
	int border;
	int count = 0;
	int thes = 20;
	if (abs(src4[index] - src1[index])<thes)
		count++;
	if (abs(src4[index] - src2[index])<thes)
		count++;
	if (abs(src4[index] - src3[index])<thes)
		count++;
	if (abs(src4[index] - src4[index])<thes)
		count++;
	if (abs(src4[index] - src5[index])<thes)
		count++;
	if (abs(src4[index] - src6[index])<thes)
		count++;
	if (abs(src4[index] - src7[index])<thes)
		count++;
	if (abs(src4[index] - src8[index])<thes)
		count++;
	if (abs(src4[index] - src9[index])<thes)
		count++;

	if (count > 7)
		pDstImage[index] = 255 - 255 / count;
	else
		pDstImage[index] = 0;

}
__global__ void cuda_sobel2D(float *src0, float *bmask, int w, int h, float *pdst, int *pPoint)
{
	//int pPoint[26] = { 846,510,866,676,747,686,951,676,776,824,976,799,863,903,761,941,985,872,834,1111,903,1122,861,1326,912,1293 };
	int point[14][2] = { 0 };
	for (int i = 0; i < 14; i++)
	{
		point[i][0] = pPoint[i * 2 + 0];
		point[i][1] = pPoint[i * 2 + 1];
	}
	int min_x = 288, min_y = 512, max_x = 0, max_y = 0;
	for (int i = 0; i < 14; i++)
	{
		if (min_x > point[i][0])
		{
			min_x = point[i][0];
		}
		if (min_y > point[i][1])
		{
			min_y = point[i][1];
		}

		if (max_x < point[i][0])
			max_x = point[i][0];

		if (max_y < point[i][1])
			max_y = point[i][1];

	}
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int index = y*w + x;
	int border;
	float tempX, tempY, magnitude;
	bool a = true;

	int weightX[9] = { -1,  0,  1 , -2,  0,  2 ,-1,  0,  1 };
	int weightY[9] = { 1,2,1,0,0,0,-1,-2,-1 };
	if (x > min_x - 20 && x < max_x + 20 && y>min_y  && y<max_y + 20)
		if (x>3 / 2 && y>3 / 2 && x < w - 3 / 2 && y < h - 3 / 2)
		{
			if (abs(src0[index] - bmask[index]) >20) {
				tempX = 0; tempY = 0;
				// make cuda code here !!!!!!!!!!!!!!!!!!!!!!!!!
				for (int i = 0; i < 3; i++)
				{
					for (int j = 0; j < 3; j++)
					{
						border = (y + i)*w + (x + j);
						tempX += (src0[border] * weightX[i * 3 + j]);
						tempY += (src0[border] * weightY[i * 3 + j]);

					}
				}
				magnitude = sqrt(tempX*tempX + tempY*tempY);
				pdst[index] = magnitude>15 ? magnitude : bmask[index];

			}
			else {
				pdst[index] = 0;
			}

		}
		else {
			pdst[index] = 0;
		}
	else
	{
		pdst[index] = 0;
	}

}
__global__ void cuda_draw(float *src0, int w, int h, float *pdst, int* pPoint)
{
	//int pPoint[26] = { 846,510,866,676,747,686,951,676,776,824,976,799,863,903,761,941,985,872,834,1111,903,1122,861,1326,912,1293 };
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int index = y*w + x;

	int point[14][2] = { 0 };
	for (int i = 0; i < 14; i++)
	{
		point[i][0] = pPoint[i * 2 + 0];
		point[i][1] = pPoint[i * 2 + 1];
	}
	int head[2] = { point[13][0],point[13][1] };
	int neck[2] = { point[12][0],point[12][1] };
	int heap[2] = { point[3][0],point[3][1] };
	int rsh[2] = { point[8][0],point[8][1] };
	int lsh[2] = { point[9][0],point[9][1] };
	int x1 = head[0];
	int y1 = head[1];
	int x2 = neck[0];
	int y2 = neck[1];
	bool k = false;
	short count = 0;
	int distance1 = (point[6][0] - point[7][0])*(point[6][0] - point[7][0]) + (point[6][1] - point[7][1])*(point[6][1] - point[7][1]);
	int d = 0;
	int d1 = (lsh[0] - rsh[0])*(lsh[0] - rsh[0]) + (lsh[1] - rsh[1])*(lsh[1] - rsh[1]);
	int d2 = (head[0] - neck[0])*(head[0] - neck[0]) + (head[1] - neck[1])*(head[1] - neck[1]);
	if (d1 > d2)
		d = d1;
	else
		d = d2;
	for (int i = 0; i < 13; i++)
	{
		if (i >= 0 && i<5) {
			x1 = point[i][0];
			y1 = point[i][1];
			x2 = point[i + 1][0];
			y2 = point[i + 1][1];
			if (isOnLine(x1, y1, x2, y2, x, y)) {
				Point ret = distance(x1, y1, x2, y2, x, y);
				int x0 = (int)ret.x;
				int y0 = (int)ret.y;
				pdst[y0*w + x0] = 255;
				if (((x0 - x)* (x0 - x) + (y0 - y)* (y0 - y)) < distance1 / 16) {
					k = true;
				}

			}
			else if (((x1 - x)* (x1 - x) + (y1 - y)* (y1 - y)) <  distance1 / 8) {
				k = true;
			}
		}
		if (i == 5)
		{
			x1 = point[i][0];
			y1 = point[i][1];
			x2 = point[i + 1][0];
			y2 = point[i + 1][1];
			if (((x1 - x)* (x1 - x) + (y1 - y)* (y1 - y)) <  distance1 / 8) {
				k = true;
			}
		}
		if (i >= 6 && i <= 10)
		{
			x1 = point[i][0];
			y1 = point[i][1];
			x2 = point[i + 1][0];
			y2 = point[i + 1][1];
			if (isOnLine(x1, y1, x2, y2, x, y)) {
				Point ret = distance(x1, y1, x2, y2, x, y);
				int x0 = (int)ret.x;
				int y0 = (int)ret.y;
				pdst[y0*w + x0] = 255;

				if (((x0 - x)* (x0 - x) + (y0 - y)* (y0 - y)) < distance1 / 16) {
					k = true;
				}

			}
			else if (((x1 - x)* (x1 - x) + (y1 - y)* (y1 - y)) <  distance1 / 16) {
				k = true;
			}
		}

		if (i == 12)
		{
			x1 = point[i][0];
			y1 = point[i][1];
			x2 = point[i + 1][0];
			y2 = point[i + 1][1];
			int x0 = (x1 + x2) / 2;
			int y0 = (y1 + y2) / 2;
			if (((x0 - x)* (x0 - x) + (y0 - y)* (y0 - y)) < ((x2 - x1)* (x2 - x1) + (y2 - y1)* (y2 - y1)) / 4) {
				k = true;
			}

		}

	}

	x1 = neck[0];
	y1 = neck[1];
	x2 = heap[0];
	y2 = heap[1];
	if (isOnLine(x1, y1, x2, y2, x, y)) {
		Point ret = distance(x1, y1, x2, y2, x, y);
		int x0 = (int)ret.x;
		int y0 = (int)ret.y;
		if (((x0 - x)* (x0 - x) + (y0 - y)* (y0 - y)) < d / 4) {
			k = true;
		}

	}
	else if (((x1 - x)* (x1 - x) + (y1 - y)* (y1 - y)) <  d / 4) {
		k = true;
	}


	if (k)
		pdst[index] = src0[index];
	else
		pdst[index] = 0;

}

void sub(float**pcuSrc, int w, int h, float *pcuDst, float *bmask, int *pPoint, float *temp)
{
	dim3 block = dim3(18, 32, 1);
	dim3 grid = dim3(w / block.x, h / block.y);
	cuda_low2D << <grid, block >> > (pcuSrc[0], pcuSrc[1], pcuSrc[2], pcuSrc[3], pcuSrc[4], pcuSrc[5], pcuSrc[6], pcuSrc[7], pcuSrc[8], pcuSrc[9], w, h, pcuDst);
	cuda_back << <grid, block >> > (pcuSrc[0], pcuSrc[1], pcuSrc[2], pcuSrc[3], pcuSrc[4], pcuSrc[5], pcuSrc[6], pcuSrc[7], pcuSrc[8], pcuSrc[9], w, h, bmask);
	cuda_sobel2D << <grid, block >> > (pcuSrc[4], bmask, w, h, temp, pPoint);
	cuda_draw << <grid, block >> > (temp, w, h, pcuDst, pPoint);
	hipDeviceSynchronize();
}