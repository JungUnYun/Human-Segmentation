#include "hip/hip_runtime.h"
// HW#7 22181250 ������
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

extern "C" void gpu_Gabor(float *pcuSrc, float *pcuDst, int w, int h, float *cuGkernel, int kernel_size);

__global__ void cuda_Filter2D(float * pSrcImage, int SrcWidth,
	int SrcHeight, float *pKernel, int KWidth, int KHeight, float *pDstImage)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int index = y*SrcWidth + x;
	int border;
	float temp;
	// 
	if (x > KWidth / 2 && y > (KHeight / 2) && x < (SrcWidth - KWidth / 2) 
		&& y < (SrcHeight - KHeight / 2))
	{
		temp = 0;
		for (int i = 0; i < KHeight; i++)
		{
			for (int j = 0; j < KWidth; j++)
			{
				border = (y + i)*SrcWidth + (x + j);
				temp += (pSrcImage[border] * pKernel[i*KWidth + j]);
			}
		}
		pDstImage[index] = temp;
//		__syncthreads();
	}
	else
	{
		pDstImage[index] = 0;
	}

}

__global__ void cuda_Shared_Filter2D(float * pSrcImage, int SrcWidth,
	int SrcHeight, float *pKernel, int KWidth, int KHeight, float *pDstImage)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int index = y*SrcWidth + x;
	int border;

	float temp;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	extern __shared__ float gmat[];

	if (x > KWidth / 2 && y >(KHeight / 2) && x < (SrcWidth - KWidth / 2)
		&& y < (SrcHeight - KHeight / 2))
	{
		temp = 0;
		if (tx < KWidth&&ty < KHeight)
		{
			// shared memory�� �������� gmat���� pKernel �ѹ��� �о� �غ���
			gmat[ty*KWidth + tx] = pKernel[ty*KWidth + tx];
		}

		for (int i = 0; i < KHeight; i++)
		{
			for (int j = 0; j < KWidth; j++)
			{
				border = (y + i)*SrcWidth + (x + j);
				temp += (pSrcImage[border] * gmat[i*KWidth + j]);
			}
		}
		pDstImage[index] = temp;
//		__syncthreads();
	}
	else
	{
		pDstImage[index] = 0;
	}

}


void gpu_Gabor(float *pcuSrc, float *pcuDst, int w, int h, float *cuGkernel, int kernel_size)
{
	// block �غ��ϰ� block�� �°� grid ����
	dim3 block = dim3(16, 16, 1); // 16  16  1 threads per block
	dim3 grid = dim3(h / block.x, w / block.y); // 32 *16 thread blocks
	
	cuda_Filter2D << < grid, block >> >(pcuSrc, w, h, cuGkernel, kernel_size, kernel_size, pcuDst);
//	cuda_Shared_Filter2D << < grid, block,sizeof(float)*kernel_size*kernel_size >> >(pcuSrc, w, h, cuGkernel, kernel_size, kernel_size, pcuDst);

	hipDeviceSynchronize();


	float *PrintKernel = new float[kernel_size*kernel_size];
	hipMemcpy(PrintKernel, cuGkernel, kernel_size*kernel_size * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i<kernel_size; i++) {
		for (int j = 0; j<kernel_size; j++)
		{
			printf("%f\t", PrintKernel[i*kernel_size + j]);
		}
		printf("\n");
	}
}
